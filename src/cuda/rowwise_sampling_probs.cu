#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <torch/script.h>

#include "../cuda_common.h"
#include "../dgs_headers.h"
#include "cub_function.h"
#include "dgs_ops.h"
#include "warpselect/WarpSelect.cuh"

#define BLOCK_SIZE 128
namespace dgs {
namespace cuda {
template <typename IdType>
inline std::pair<torch::Tensor, torch::Tensor> _GetSubAndTempIndptr(
    torch::Tensor seeds, torch::Tensor indptr, int64_t num_pick, bool replace) {
  int64_t num_items = seeds.numel();
  torch::Tensor sub_indptr = torch::empty((num_items + 1), indptr.options());
  torch::Tensor temp_indptr = torch::empty((num_items + 1), indptr.options());
  thrust::device_ptr<IdType> sub_prefix(
      static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
  thrust::device_ptr<IdType> temp_prefix(
      static_cast<IdType *>(temp_indptr.data_ptr<IdType>()));

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(num_items),
      [in = seeds.data_ptr<IdType>(), in_indptr = indptr.data_ptr<IdType>(),
       sub_ptr = thrust::raw_pointer_cast(sub_prefix),
       tmp_ptr = thrust::raw_pointer_cast(temp_prefix), replace, num_pick,
       num_items] __device__(int i) mutable {
        IdType row = in[i];
        IdType begin = in_indptr[row];
        IdType end = in_indptr[row + 1];
        IdType deg = end - begin;
        if (replace) {
          sub_ptr[i] = deg == 0 ? 0 : num_pick;
          tmp_ptr[i] = deg;
        } else {
          sub_ptr[i] = MIN(deg, num_pick);
          tmp_ptr[i] = deg > num_pick ? deg : 0;
        }
        if (i == num_items - 1) {
          sub_ptr[num_items] = 0;
          tmp_ptr[num_items] = 0;
        }
      });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(sub_prefix), num_items + 1);
  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(temp_prefix),
                           num_items + 1);
  return {sub_indptr, temp_indptr};
}

template <typename IdType, typename FloatType, int TILE_SIZE, int BLOCK_WARPS,
          int WARP_SIZE, int NumWarpQ, int NumThreadQ>
__global__ void _CSRRowWiseSampleKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_ptr,
    const IdType *__restrict__ const in_cols,
    const FloatType *__restrict__ const prob,
    const IdType *__restrict__ const out_ptr,
    IdType *__restrict__ const out_rows, IdType *const out_cols) {
  // we assign one warp per row
  assert(num_picks <= 32);
  assert(blockDim.x == WARP_SIZE);
  assert(blockDim.y == BLOCK_WARPS);

  __shared__ IdType warpselect_out_index[WARP_SIZE * BLOCK_WARPS];

  // init warpselect
  warpselect::WarpSelect<FloatType, IdType,
                         true,  // produce largest values
                         warpselect::Comparator<FloatType>, NumWarpQ,
                         NumThreadQ, WARP_SIZE * BLOCK_WARPS>
      heap(warpselect::_Limits<FloatType>::getMin(), -1, num_picks);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      MIN(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
              threadIdx.y * WARP_SIZE + threadIdx.x, 0, &rng);

  int laneid = threadIdx.x % WARP_SIZE;
  int warp_id = threadIdx.y;
  IdType *warpselect_out_index_per_warp =
      warpselect_out_index + warp_id * WARP_SIZE;

  while (out_row < last_row) {
    const IdType row = in_rows[out_row];
    const IdType in_row_start = in_ptr[row];
    const IdType deg = in_ptr[row + 1] - in_row_start;
    const IdType out_row_start = out_ptr[out_row];
    // A-Res value needs to be calculated only if deg is greater than num_picks
    // in weighted rowwise sampling without replacement
    if (deg > num_picks) {
      heap.reset();
      int limit = warpselect::roundDown(deg, WARP_SIZE);
      IdType i = laneid;

      for (; i < limit; i += WARP_SIZE) {
        FloatType item_prob = prob[in_row_start + i];
        FloatType ares_prob = __powf(hiprand_uniform(&rng), 1.0f / item_prob);
        heap.add(ares_prob, i);
      }

      if (i < deg) {
        FloatType item_prob = prob[in_row_start + i];
        FloatType ares_prob = __powf(hiprand_uniform(&rng), 1.0f / item_prob);
        heap.addThreadQ(ares_prob, i);
        i += WARP_SIZE;
      }

      heap.reduce();
      heap.writeOutV(warpselect_out_index_per_warp, num_picks);

      for (int idx = laneid; idx < num_picks; idx += WARP_SIZE) {
        const IdType out_idx = out_row_start + idx;
        const IdType in_idx = warpselect_out_index_per_warp[idx] + in_row_start;
        out_rows[out_idx] = static_cast<IdType>(row);
        out_cols[out_idx] = in_cols[in_idx];
      }
    } else {
      for (int idx = threadIdx.x; idx < deg; idx += WARP_SIZE) {
        // get in and out index
        const IdType out_idx = out_row_start + idx;
        const IdType in_idx = in_row_start + idx;
        // copy permutation over
        out_rows[out_idx] = static_cast<IdType>(row);
        out_cols[out_idx] = in_cols[in_idx];
      }
    }

    out_row += BLOCK_WARPS;
  }
}

template <typename IdType, typename FloatType, int TILE_SIZE, int BLOCK_WARPS,
          int WARP_SIZE>
__global__ void _CSRRowWiseSampleReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_ptr,
    const IdType *__restrict__ const in_cols,
    const FloatType *__restrict__ const prob,
    const IdType *__restrict__ const out_ptr,
    const IdType *__restrict__ const cdf_ptr, FloatType *__restrict__ const cdf,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == WARP_SIZE);
  assert(blockDim.y == BLOCK_WARPS);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      MIN(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
              threadIdx.y * BLOCK_WARPS + threadIdx.x, 0, &rng);

  typedef hipcub::WarpScan<FloatType> WarpScan;
  __shared__ typename WarpScan::TempStorage temp_storage[BLOCK_WARPS];
  int warp_id = threadIdx.y;
  int laneid = threadIdx.x;

  while (out_row < last_row) {
    const IdType row = in_rows[out_row];
    const IdType in_row_start = in_ptr[row];
    const IdType out_row_start = out_ptr[out_row];
    const IdType cdf_row_start = cdf_ptr[out_row];
    const IdType deg = in_ptr[row + 1] - in_row_start;
    const FloatType MIN_THREAD_DATA = static_cast<FloatType>(0.0f);

    if (deg > 0) {
      IdType max_iter = (1 + (deg - 1) / WARP_SIZE) * WARP_SIZE;
      // Have the block iterate over segments of items

      FloatType warp_aggregate = static_cast<FloatType>(0.0f);
      for (int idx = laneid; idx < max_iter; idx += WARP_SIZE) {
        FloatType thread_data =
            idx < deg ? prob[in_row_start + idx] : MIN_THREAD_DATA;
        if (laneid == 0) thread_data += warp_aggregate;
        thread_data = max(thread_data, MIN_THREAD_DATA);

        WarpScan(temp_storage[warp_id])
            .InclusiveSum(thread_data, thread_data, warp_aggregate);
        __syncwarp();
        // Store scanned items to cdf array
        if (idx < deg) {
          cdf[cdf_row_start + idx] = thread_data;
        }
      }
      __syncwarp();

      for (int idx = laneid; idx < num_picks; idx += WARP_SIZE) {
        // get random value
        FloatType sum = cdf[cdf_row_start + deg - 1];
        FloatType rand = static_cast<FloatType>(hiprand_uniform(&rng) * sum);
        // get the offset of the first value within cdf array which is greater
        // than random value.
        IdType item = hipcub::UpperBound<FloatType *, IdType, FloatType>(
            &cdf[cdf_row_start], deg, rand);
        item = MIN(item, deg - 1);
        // get in and out index
        const IdType in_idx = in_row_start + item;
        const IdType out_idx = out_row_start + idx;
        // copy permutation over
        out_rows[out_idx] = static_cast<IdType>(row);
        out_cols[out_idx] = in_cols[in_idx];
      }
    }
    out_row += BLOCK_WARPS;
  }
}

template <typename IdType, typename FloatType>
std::tuple<torch::Tensor, torch::Tensor> RowWiseSamplingProbCUDA(
    torch::Tensor seeds, torch::Tensor indptr, torch::Tensor indices,
    torch::Tensor probs, int64_t num_picks, bool replace) {
  int num_rows = seeds.numel();
  torch::Tensor sub_indptr, temp_indptr;
  std::tie(sub_indptr, temp_indptr) =
      _GetSubAndTempIndptr<IdType>(seeds, indptr, num_picks, replace);
  thrust::device_ptr<IdType> sub_prefix(
      static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
  thrust::device_ptr<IdType> temp_prefix(
      static_cast<IdType *>(temp_indptr.data_ptr<IdType>()));
  int nnz = sub_prefix[num_rows];
  int temp_size = temp_prefix[num_rows];

  torch::Tensor coo_row = torch::empty(nnz, seeds.options());
  torch::Tensor coo_col = torch::empty(nnz, indices.options());
  torch::Tensor temp = torch::empty(temp_size, probs.options());

  const uint64_t random_seed = 7777;
  constexpr int WARP_SIZE = 32;
  constexpr int BLOCK_WARPS = BLOCK_SIZE / WARP_SIZE;
  // constexpr int TILE_SIZE = BLOCK_WARPS * 16;
  constexpr int TILE_SIZE = 16;
  // constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
  if (replace) {
    const dim3 block(WARP_SIZE, BLOCK_WARPS);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    _CSRRowWiseSampleReplaceKernel<IdType, FloatType, TILE_SIZE, BLOCK_WARPS,
                                   WARP_SIZE><<<grid, block>>>(
        random_seed, num_picks, num_rows, seeds.data_ptr<IdType>(),
        indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
        probs.data_ptr<FloatType>(), sub_indptr.data_ptr<IdType>(),
        temp_indptr.data_ptr<IdType>(), temp.data_ptr<FloatType>(),
        coo_row.data_ptr<IdType>(), coo_col.data_ptr<IdType>());
  } else {
    const dim3 block(WARP_SIZE, BLOCK_WARPS);
    const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
    _CSRRowWiseSampleKernel<IdType, FloatType, TILE_SIZE, BLOCK_WARPS,
                            WARP_SIZE, 32, 2><<<grid, block>>>(
        random_seed, num_picks, num_rows, seeds.data_ptr<IdType>(),
        indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
        probs.data_ptr<FloatType>(), sub_indptr.data_ptr<IdType>(),
        coo_row.data_ptr<IdType>(), coo_col.data_ptr<IdType>());
  }

  return std::make_tuple(coo_row, coo_col);
}

std::tuple<torch::Tensor, torch::Tensor> RowWiseSamplingProbCUDA(
    torch::Tensor seeds, torch::Tensor indptr, torch::Tensor indices,
    torch::Tensor probs, int64_t num_picks, bool replace) {
  CHECK_CUDA(seeds);
  CHECK_CUDA(indptr);
  CHECK_CUDA(indices);
  CHECK_CUDA(probs);
  DGS_ID_TYPE_SWITCH(indptr.dtype(), IdType, {
    DGS_VALUE_TYPE_SWITCH(probs.dtype(), FloatType, {
      return RowWiseSamplingProbCUDA<IdType, FloatType>(
          seeds, indptr, indices, probs, num_picks, replace);
    });
  });

  return std::make_tuple(torch::Tensor(), torch::Tensor());
}
}  // namespace cuda
}  // namespace dgs