#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <torch/script.h>

#include "atomic.h"
#include "cub_function.h"
#include "cuda_common.h"
#include "dgs_ops.h"

#define BLOCK_SIZE 128

namespace dgs {

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    chunk_tensor_wrapper<IdType> *__restrict__ in_index,
    const IdType *__restrict__ const out_ptr,
    const IdType *__restrict__ const row_begin,
    const IdType *__restrict__ const row_end,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = row_begin[out_row];
    const int64_t deg = row_end[out_row] - in_row_start;
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index->At(in_idx);
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_cols[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          atomic::AtomicMax(out_cols + out_row_start + num, IdType(idx));
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_cols[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index->At(perm_idx);
      }
    }
    out_row += 1;
  }
}

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    chunk_tensor_wrapper<IdType> *__restrict__ in_index,
    const IdType *__restrict__ const out_ptr,
    const IdType *__restrict__ const row_begin,
    const IdType *__restrict__ const row_end,
    IdType *__restrict__ const out_rows, IdType *const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = row_begin[out_row];
    const int64_t out_row_start = out_ptr[out_row];
    const int64_t deg = row_end[out_row] - in_row_start;

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        out_cols[out_idx] = in_index->At(in_row_start + edge);
      }
    }
    out_row += 1;
  }
}

template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor>
RowWiseSamplingUniformCUDAWithChunkTensorCUDA(
    torch::Tensor seeds, c10::intrusive_ptr<ChunkTensor> indptr,
    c10::intrusive_ptr<ChunkTensor> indices, int64_t num_picks, bool replace) {
  CHECK_CUDA(seeds);
  chunk_tensor_wrapper<IdType> *d_indptr_wrapper_ptr =
      reinterpret_cast<chunk_tensor_wrapper<IdType> *>(
          indptr->wrapper_chunktensor_ptr_);
  chunk_tensor_wrapper<IdType> *d_indices_wrapper_ptr =
      reinterpret_cast<chunk_tensor_wrapper<IdType> *>(
          indices->wrapper_chunktensor_ptr_);

  int num_items = seeds.numel();
  torch::Tensor row_begin_tensor = torch::empty(
      num_items,
      torch::TensorOptions().dtype(indptr->dtype_).device(torch::kCUDA));
  torch::Tensor row_end_tensor = torch::empty(
      num_items,
      torch::TensorOptions().dtype(indptr->dtype_).device(torch::kCUDA));
  torch::Tensor sub_indptr = torch::empty(
      (num_items + 1),
      torch::TensorOptions().dtype(indptr->dtype_).device(torch::kCUDA));
  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(num_items),
      [in = seeds.data_ptr<IdType>(), in_indptr = d_indptr_wrapper_ptr,
       out = sub_indptr.data_ptr<IdType>(), replace, num_picks,
       row_begin = row_begin_tensor.data_ptr<IdType>(),
       row_end = row_end_tensor.data_ptr<IdType>()] __device__(int i) mutable {
        IdType row = in[i];
        row_begin[i] = in_indptr->At(row);
        row_end[i] = in_indptr->At(row + 1);
        if (replace) {
          out[i] = (row_end[i] - row_begin[i]) == 0 ? 0 : num_picks;
        } else {
          out[i] = MIN(row_end[i] - row_begin[i], num_picks);
        }
      });

  cub_exclusiveSum<IdType>(sub_indptr.data_ptr<IdType>(), num_items + 1);
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
  int nnz = item_prefix[num_items];

  torch::Tensor coo_row = torch::empty(nnz, seeds.options());
  torch::Tensor coo_col = torch::empty(
      nnz, torch::TensorOptions().dtype(indices->dtype_).device(torch::kCUDA));

  const uint64_t random_seed = 7777;
  constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
  if (replace) {
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_items + TILE_SIZE - 1) / TILE_SIZE);
    _CSRRowWiseSampleUniformReplaceKernel<IdType, TILE_SIZE><<<grid, block>>>(
        random_seed, num_picks, num_items, seeds.data_ptr<IdType>(),
        d_indices_wrapper_ptr, sub_indptr.data_ptr<IdType>(),
        row_begin_tensor.data_ptr<IdType>(), row_end_tensor.data_ptr<IdType>(),
        coo_row.data_ptr<IdType>(), coo_col.data_ptr<IdType>());
  } else {
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_items + TILE_SIZE - 1) / TILE_SIZE);
    _CSRRowWiseSampleUniformKernel<IdType, TILE_SIZE><<<grid, block>>>(
        random_seed, num_picks, num_items, seeds.data_ptr<IdType>(),
        d_indices_wrapper_ptr, sub_indptr.data_ptr<IdType>(),
        row_begin_tensor.data_ptr<IdType>(), row_end_tensor.data_ptr<IdType>(),
        coo_row.data_ptr<IdType>(), coo_col.data_ptr<IdType>());
  }

  return std::make_tuple(coo_row, coo_col);
}

std::tuple<torch::Tensor, torch::Tensor> RowWiseSamplingUniformWithChunkTensor(
    torch::Tensor seeds, c10::intrusive_ptr<ChunkTensor> indptr,
    c10::intrusive_ptr<ChunkTensor> indices, int64_t num_picks, bool replace) {
  DGS_ID_TYPE_SWITCH(indptr->dtype_, IdType, {
    return RowWiseSamplingUniformCUDAWithChunkTensorCUDA<IdType>(
        seeds, indptr, indices, num_picks, replace);
  });
  return std::make_tuple(torch::Tensor(), torch::Tensor());
}
}  // namespace dgs